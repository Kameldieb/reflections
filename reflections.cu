#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <vector>
#include <Eigen/Dense>
#include <X11/Xlib.h>
#include "X11/keysym.h"
#include <unistd.h>
#include <chrono>


#define SCREEN_WIDTH 1.25
#define SCREEN_HEIGHT 0.7

// used in update_orientation
#define ANG1 0.3
#define ANG2 0.2
#define SPEED 5
#define TSTEP 0.02   // * 10^-6s
#define ANG1_RESET 1

#define EPS 0.00001   // for division by "zero"
#define MAX_REFLECTIONS 10
#define MAX_BALLS 1000    // max number of balls
#define MAX_CYLS 1000


typedef Eigen::Vector3f Vec;

using namespace std;

class Point {
public:
  float x, y, z;
  void rotate(Point axis, float amount) {
    // rotates point around axis by amount (in radians)


  }
};


class Ball {
public:
  Vec center;
  float radius;
};


class Cylinder {
public:
  // cylinder axis between base and base+dir
  Vec base;
  Vec dir;
  float radius;

};


// copied from somewhere
bool key_is_pressed(KeySym ks) {
    Display *dpy = XOpenDisplay(":1");
    char keys_return[32];
    XQueryKeymap(dpy, keys_return);
    KeyCode kc2 = XKeysymToKeycode(dpy, ks);
    bool isPressed = !!(keys_return[kc2 >> 3] & (1 << (kc2 & 7)));
    XCloseDisplay(dpy);
    return isPressed;
}


void update_orientation(Eigen::Matrix3f *dir) {
    if (key_is_pressed(XK_Left)) {
      *dir = Eigen::AngleAxisf(-ANG1*TSTEP*M_PI, dir->col(0)) * *dir;
      *dir = Eigen::AngleAxisf(0.5*ANG1*TSTEP*M_PI, dir->col(2)) * *dir;
      *dir = Eigen::AngleAxisf(-0.4*ANG1*TSTEP*M_PI, dir->col(1)) * *dir;
    }
    if (key_is_pressed(XK_Right)) {
      *dir = Eigen::AngleAxisf(ANG1*TSTEP*M_PI, dir->col(0)) * *dir;
      *dir = Eigen::AngleAxisf(-0.5*ANG1*TSTEP*M_PI, dir->col(2)) * *dir;
      *dir = Eigen::AngleAxisf(-0.4*ANG1*TSTEP*M_PI, dir->col(1)) * *dir;
    }
    if (key_is_pressed(XK_Up)) {
      *dir = Eigen::AngleAxisf(ANG2*TSTEP*M_PI, dir->col(1)) * *dir;
    }
    if (key_is_pressed(XK_Down)) {
      *dir = Eigen::AngleAxisf(-ANG2*TSTEP*M_PI, dir->col(1)) * *dir;
    }
    float roll = dir->col(1).z();
    *dir = Eigen::AngleAxisf(-ANG1_RESET*TSTEP*roll, dir->col(0)) * *dir;
}


void update_pos(Vec *pos, Eigen::Matrix3f *dir) {
  *pos += dir->col(0)*TSTEP*SPEED;
}


Vec **init_ray_array(int height, int width) {
  Vec **res;
  hipMallocManaged(&res, sizeof(Vec*)*height);
  for (int i = 0; i < height; ++i) {
    hipMallocManaged(&res[i], sizeof(Vec)*width);
  }
  return res;
}


char **init_pic_array(int height, int width) {
  char **res;
  hipMallocManaged(&res, height*sizeof(char*));
  for (int i = 0; i < height; ++i) {
    hipMallocManaged(&res[i], (width+1)*sizeof(char));
    res[i][width] = 0;
  }
  return res;
}


__global__
void create_rays(Vec **rays, Eigen::Matrix3f *dir, int height, int width) {
  // rays go through "pixels" of "screen" in front of the plane, written into rays array
  int index = threadIdx.x;
  int stride = blockDim.x;
  Vec upper_left = dir->col(0) + SCREEN_WIDTH/2.0*dir->col(1) + SCREEN_HEIGHT/2.0*dir->col(2);
  for (int i = blockIdx.x; i < height; i += gridDim.x) {
    for (int j = index; j < width; j += stride) {
      rays[i][j] = upper_left - SCREEN_HEIGHT*(float)i/(height-1) * dir->col(2) - SCREEN_WIDTH*(float)j/(width-1) * dir->col(1);
      rays[i][j].normalize();
    }
  }
}


__device__
bool check_reflection_ball(Ball &ball, Vec &from, Vec &ray, float &min_dist_to_refl, Vec &reflection_point, Vec &reflection_plane) {
  // checks if from + t*ray intersects ball, if so checks if intersection point is closer to from than min_dist_to_refl
  // if that is the case, write reflection_point and reflection_plane, return true
  // else return false
  // math: use abc formula to solve for intersections, then use closer intersection point
  Vec tmp = from - ball.center;
  float b = 2*ray.dot(tmp);
  float c = tmp.dot(tmp) - ball.radius*ball.radius;
  float under_root = b*b - 4*c;
  if (under_root < 0) return false;   // no reflection

  float t = (- b - sqrt(under_root))/2;    // only the smaller solution is useful
  if (t < 0) return false;    // reflection behind viewer
  // t > 0, thus intersection is at pos + t*ray
  if (t < min_dist_to_refl) {
    reflection_point = from + t * ray;
    if (reflection_point.z() < 0) return false;
    reflection_plane = reflection_point - ball.center;
    min_dist_to_refl = t;
    return true;
  }
  return false;
}

__device__
bool check_reflection_cylinder(Cylinder &cyl, Vec &from, Vec &ray, float &min_dist_to_refl, Vec &reflection_point, Vec &reflection_plane) {
  // does exactly the same as check_reflection_ball, except with a cylinder
  // the math is more complicated, rough outline:
  // first check if minimal distance of ray to cylinder axis is smaller than cylinder radius
  // if so, calculate first intersection using the minimal distance point
  // then check if this intersection point is still in the bounds of the cylinder
  // not even sure if my math was correct, they sometimes look weird but most of the time ok

  // cylinder: a+tb, ray: c+sd
  Vec a, b, c, d, p, q, r, u;
  float s1, s2, t1, t2, R;

  a = cyl.base;
  b = cyl.dir;
  c = from;
  d = ray;
  R = cyl.radius;

  float dnom = b.dot(d) * b.dot(d) - b.dot(b);
  if (abs(dnom) < EPS) return false;  // cylinder axis parallel to ray
  s1 = (d.dot(c-a) * b.dot(b) - b.dot(c-a) * b.dot(d))/dnom;
  if (s1 < 0) return false; // point of minimal distance is behind viewer, thus first point of intersection as well
  t1 = (s1 * b.dot(d) + b.dot(c-a))/b.dot(b);
  p = c + s1 * d;   // point on ray with minimal distance to cylinder axis
  q = p - (a + t1 * b);        // vector from cylinder axis to p, orthogonal to cyl axis
  if (q.dot(q) >= R*R) return false;   // no reflection
  s2 = sqrt((R*R - q.dot(q))/(1-d.dot(b)/b.dot(b)));  // s2 is how far to go back from p on the ray to get to an intersection point
  if (s2 > s1) return false;  // first point of intersection is behind viewer
  if (s1-s2 > min_dist_to_refl) return false;
  reflection_point = p - s2*d;
  if (reflection_point.z() < 0) return false;
  u = reflection_point - a;
  reflection_plane = u - u.dot(b)/b.dot(b)*b;   // projected u onto cyl axis giving plane of reflection 

  // check if reflection point is in the cylinders bounds
  r = reflection_point - reflection_plane - a;  // vector from a to projection of reflection_point onto cyl axis
  if (b.x() != 0) t2 = r.x()/b.x();
  else if (b.y() != 0) t2 = r.y()/b.y();
  else if (b.z() != 0) t2 = r.z()/b.z();
  else return false;
  if (t2 >= 0 && t2 <= 1) {
    min_dist_to_refl = s1-s2;
    return true;
  }
  else return false;
}


__global__
void calc_rays(Vec *pos, Vec **rays, char **pic, Ball *balls, int num_balls, Cylinder *cyls, int num_cyls, int height, int width) {
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = blockIdx.x; i < height; i += gridDim.x) {
    for (int j = index; j < width; j += stride) {
      Vec from = *pos;
      Vec ray = rays[i][j];
      bool ever_reflected = false;
      bool reflected_now;
      Vec reflection_point;
      Vec reflection_plane;

      // do up to 10 ball reflections
      for (int k = 0; k < MAX_REFLECTIONS; ++k) {
        reflected_now = false;
        float min_dist_to_refl = 1000000;   // initialize to large distance

        // check ball reflections
        for (int ball_i = 0; ball_i < num_balls; ++ball_i) {
          if (check_reflection_ball(balls[ball_i], from, ray, min_dist_to_refl, reflection_point, reflection_plane)) {
            reflected_now = true;
          }
        }
        // and cylinder reflections
        for (int cyl_i = 0; cyl_i < num_cyls; ++cyl_i) {
          if (check_reflection_cylinder(cyls[cyl_i], from, ray, min_dist_to_refl, reflection_point, reflection_plane)) {
            reflected_now = true;
          }
        }
        if (!reflected_now) break;   // no reflection, thus no more checking necessary
        // reflect off of plane given by reflection_point and reflection_plane
        // math done on paper
        ray -= 2*ray.dot(reflection_plane)/reflection_plane.dot(reflection_plane) * reflection_plane;
        from = reflection_point + 0.001*reflection_plane;   // new start position for the ray, added a bit of refl to get out of the ball
        ever_reflected = true;
      }

      // check if it hits floor, color appropriately (using color code and string, change string to change looks)
      int color = 0;
      char code_to_char[7] = ". ;,EL";
      if (ever_reflected) color++;
      if (ray.z() < 0) {
        from -= from.z()/ray.z() * ray;
        if ((int)(floor(from.x()) + floor(from.y())) % 2 == 0) {
          // dark tiles
          color += 4;
        }
        else {
          // light tiles
          color += 2;
        }
      }
      pic[i][j] = code_to_char[color];
    }
  }
}


void draw_pic(char **pic, int height, int width) {
  cout << "\033[0;0f";
  for (int i = 0; i < height; ++i) {
    cout << pic[i];
    cout << '\n';
  }
  cout << '\n';
}


int main(int argc, char *argv[]) {
  int height, width;
  if (argc == 1) {
    // no window sizes given, defaults to 200x100
    height = 100;
    width = 200;
  }
  else {
    height = stoi(argv[2]);
    width = stoi(argv[1]);
  }

  Vec *pos;
  hipMallocManaged(&pos, sizeof(Vec));
  *pos = {-40, -2, 8};
  // orientation is stored as a matrix:
  // from the planes perspective:
  // first column vector points forward
  // second one points to the left 
  // third one points up
  Eigen::Matrix3f *dir;
  hipMallocManaged(&dir, sizeof(Eigen::Matrix3f));
  *dir << 1, 0, 0, 0, 1, 0, 0, 0, 1;

  int num_balls = 0;
  int num_cyls = 0;
  Ball *balls;
  hipMallocManaged(&balls, MAX_BALLS*sizeof(Ball));
  Cylinder *cyls;
  hipMallocManaged(&cyls, MAX_CYLS*sizeof(Cylinder));

  // create some balls and cylinders
  for (int i = 0; i < 5; ++i) {
    cyls[num_cyls++] = (Cylinder) {(Vec) {7*i, -7, -2}, (Vec) {0, 20, 40}, 2};
    cyls[num_cyls++] = (Cylinder) {(Vec) {7*i + 3, 7, -2}, (Vec) {0, -20, 40}, 2};
  }

  cyls[num_cyls++] = (Cylinder) {(Vec) {80, 20, 0}, (Vec) {0, 0, 10}, 3};
  balls[num_balls++] = (Ball) {(Vec) {80, 20, 10}, 5};

  balls[num_balls++] = (Ball) {(Vec) {70, 70, 8}, 8};;
  balls[num_balls++] = (Ball) {(Vec) {70, 70, 22}, 6};;
  balls[num_balls++] = (Ball) {(Vec) {70, 70, 32}, 4};;

  for (int i = 1; i < 10; ++i) {
    balls[num_balls++] = (Ball) {(Vec) {20, 70, i*i}, i};;
  }
  for (int i = 1; i < 15; ++i) {
    balls[num_balls++] = (Ball) {(Vec) {-i, -30-(i+1)*(i+1), i}, i};
    cyls[num_cyls++] = (Cylinder) {(Vec) {5+i, -30-(i+1)*(i+1), -i/3}, (Vec) {0, 10+2*i, 40+8*i}, i};
  }

  // initialize stuff
  Vec **rays = init_ray_array(height, width);  // holds directions of the rays (later)
  char **pic = init_pic_array(height, width);

  int blockSize = 512;
  int numBlocks = 256;


  // main loop:
  while(1) {
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    
    update_orientation(dir);
    update_pos(pos, dir);
    create_rays<<<numBlocks, blockSize>>>(rays, dir, height, width);
    hipDeviceSynchronize();
    calc_rays<<<numBlocks, blockSize>>>(pos, rays, pic, balls, num_balls, cyls, num_cyls, height, width);
    hipDeviceSynchronize();
    draw_pic(pic, height, width);

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    int elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
    if (TSTEP*1000000 > elapsed) usleep(TSTEP*1000000 - elapsed);
  }
}